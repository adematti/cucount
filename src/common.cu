#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "common.h"


// Wrapper for calloc with error handling
void* my_calloc(size_t num, size_t size) {
    void* ptr = calloc(num, size);
    if (!ptr) {
        log_message(LOG_LEVEL_ERROR, "Memory allocation failed in my_calloc for %zu elements of size %zu.\n", num, size);
        exit(EXIT_FAILURE); // Exit the program on allocation failure
    }
    return ptr;
}

// Wrapper for malloc with error handling
void* my_malloc(size_t size) {
    void* ptr = malloc(size);
    if (!ptr) {
        log_message(LOG_LEVEL_ERROR, "Memory allocation failed in my_malloc for size %zu.\n", size);
        exit(EXIT_FAILURE); // Exit the program on allocation failure
    }
    return ptr;
}

void copy_particles_to_device(Particles particles, Particles *device_particles, bool struct_only) {
    CUDA_CHECK(hipMalloc((void**) device_particles, sizeof(Particles)));
    CUDA_CHECK(hipMemcpy(device_particles, &particles, sizeof(Particles), hipMemcpyHostToDevice));
    if (struct_only) {
        device_particles->positions = particles.positions;
        device_particles->weights = particles.weights;
    }
    else {
        CUDA_CHECK(hipMalloc((void **) &(device_particles->positions), NDIM * particles.size * sizeof(FLOAT)));
        CUDA_CHECK(hipMemcpy(device_particles->positions, particles.positions, NDIM * particles.size * sizeof(FLOAT), hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &(device_particles->weights), mesh.total_nparticles * sizeof(FLOAT)));
        CUDA_CHECK(hipMemcpy(device_particles->weights, particles.weights, particles.size * sizeof(FLOAT), hipMemcpyHostToDevice));
    }
}


void copy_mesh_to_device(Mesh mesh, Mesh *device_mesh, bool struct_only) {
    CUDA_CHECK(hipMalloc((void**) device_mesh, sizeof(Mesh)));
    CUDA_CHECK(hipMemcpy(device_mesh, &mesh, sizeof(Mesh), hipMemcpyHostToDevice));
    if (struct_only) {
        device_mesh->nparticles = mesh.nparticles;
        device_mesh->cumnparticles = mesh.cumnparticles;
        device_mesh->spositions = mesh.spositions;
        device_mesh->positions = mesh.positions;
        device_mesh->weights = mesh.weights;
    }
    else {
        CUDA_CHECK(hipMalloc((void **) &(device_mesh->nparticles), mesh.size * sizeof(size_t)));
        CUDA_CHECK(hipMemcpy(device_mesh->nparticles, mesh.nparticles, mesh.size * sizeof(size_t), hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &(device_mesh->cumnparticles), mesh.size * sizeof(size_t)));
        CUDA_CHECK(hipMemcpy(device_mesh->cumnparticles, mesh.cumnparticles, mesh.size * sizeof(size_t), hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &(device_mesh->spositions), NDIM * mesh.total_nparticles * sizeof(FLOAT)));
        CUDA_CHECK(hipMemcpy(device_mesh->spositions, mesh.spositions, NDIM * mesh.total_nparticles * sizeof(FLOAT), hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &(device_mesh->positions), NDIM * mesh.total_nparticles * sizeof(FLOAT)));
        CUDA_CHECK(hipMemcpy(device_mesh->positions, mesh.positions, NDIM * mesh.total_nparticles * sizeof(FLOAT), hipMemcpyHostToDevice));

        CUDA_CHECK(hipMalloc((void **) &(device_mesh->weights), mesh.total_nparticles * sizeof(FLOAT)));
        CUDA_CHECK(hipMemcpy(device_mesh->weights, mesh.weights, mesh.total_nparticles * sizeof(FLOAT), hipMemcpyHostToDevice));
    }
}


void free_device_particles(Particles *particles) {
    // Free GPU memory
    CUDA_CHECK(hipFree(particles->positions));
    CUDA_CHECK(hipFree(particles->weights));
}

void free_device_mesh(Mesh *mesh) {
    // Free GPU memory
    CUDA_CHECK(hipFree(mesh->nparticles));
    CUDA_CHECK(hipFree(mesh->cumnparticles));
    CUDA_CHECK(hipFree(mesh->spositions));
    CUDA_CHECK(hipFree(mesh->positions));
    CUDA_CHECK(hipFree(mesh->weights));
}


// Global variables for block and thread configuration
int nblocks = 0;
int nthreads_per_block = 0;

// Function to determine block and thread configuration
void configure_cuda_kernel(void (*kernel)(void)) {
    if (nthreads_per_block <= 0) {
        hipOccupancyMaxPotentialBlockSize(
            &nblocks,
            &nthreads_per_block,
            kernel,
            0,
            0
        );
    }
    log_message(LOG_LEVEL_INFO, "Configured kernel with %d blocks and %d threads per block.\n", nblocks, nthreads_per_block);
}